#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "IntegratorHPMCMonoGPU.cuh"
#include "hoomd/RandomNumbers.h"

namespace hpmc
{
namespace gpu
{
namespace kernel
{

//! Kernel to generate expanded cells
/*! \param d_excell_idx Output array to list the particle indices in the expanded cells
    \param d_excell_size Output array to list the number of particles in each expanded cell
    \param excli Indexer for the expanded cells
    \param d_cell_idx Particle indices in the normal cells
    \param d_cell_size Number of particles in each cell
    \param d_cell_adj Cell adjacency list
    \param ci Cell indexer
    \param cli Cell list indexer
    \param cadji Cell adjacency indexer
    \param ngpu Number of active devices

    gpu_hpmc_excell_kernel executes one thread per cell. It gathers the particle indices from all neighboring cells
    into the output expanded cell.
*/
__global__ void hpmc_excell(unsigned int *d_excell_idx,
                            unsigned int *d_excell_size,
                            const Index2D excli,
                            const unsigned int *d_cell_idx,
                            const unsigned int *d_cell_size,
                            const unsigned int *d_cell_adj,
                            const Index3D ci,
                            const Index2D cli,
                            const Index2D cadji,
                            const unsigned int ngpu)
    {
    // compute the output cell
    unsigned int my_cell = 0;
    my_cell = blockDim.x * blockIdx.x + threadIdx.x;

    if (my_cell >= ci.getNumElements())
        return;

    unsigned int my_cell_size = 0;

    // loop over neighboring cells and build up the expanded cell list
    for (unsigned int offset = 0; offset < cadji.getW(); offset++)
        {
        unsigned int neigh_cell = d_cell_adj[cadji(offset, my_cell)];

        // iterate over per-device cell lists
        for (unsigned int igpu = 0; igpu < ngpu; ++igpu)
            {
            unsigned int neigh_cell_size = d_cell_size[neigh_cell+igpu*ci.getNumElements()];

            for (unsigned int k = 0; k < neigh_cell_size; k++)
                {
                // read in the index of the new particle to add to our cell
                unsigned int new_idx = d_cell_idx[cli(k, neigh_cell)+igpu*cli.getNumElements()];
                d_excell_idx[excli(my_cell_size, my_cell)] = new_idx;
                my_cell_size++;
                }
            }
        }

    // write out the final size
    d_excell_size[my_cell] = my_cell_size;
    }

//! Kernel for grid shift
/*! \param d_postype postype of each particle
    \param d_image Image flags for each particle
    \param N number of particles
    \param box Simulation box
    \param shift Vector by which to translate the particles

    Shift all the particles by a given vector.

    \ingroup hpmc_kernels
*/
__global__ void hpmc_shift(Scalar4 *d_postype,
                          int3 *d_image,
                          const unsigned int N,
                          const BoxDim box,
                          const Scalar3 shift)
    {
    // identify the active cell that this thread handles
    unsigned int my_pidx = blockIdx.x * blockDim.x + threadIdx.x;

    // this thread is inactive if it indexes past the end of the particle list
    if (my_pidx >= N)
        return;

    // pull in the current position
    Scalar4 postype = d_postype[my_pidx];

    // shift the position
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    pos += shift;

    // wrap the particle back into the box
    int3 image = d_image[my_pidx];
    box.wrap(pos, image);

    // write out the new position and orientation
    d_postype[my_pidx] = make_scalar4(pos.x, pos.y, pos.z, postype.w);
    d_image[my_pidx] = image;
    }

//!< Kernel to accept/reject
__global__ void hpmc_accept(const unsigned int *d_update_order_by_ptl,
                 const unsigned int *d_trial_move_type,
                 const unsigned int *d_reject_out_of_cell,
                 unsigned int *d_reject,
                 unsigned int *d_reject_out,
                 const unsigned int *d_nneigh,
                 const unsigned int *d_nlist,
                 const unsigned int N_old,
                 const unsigned int N,
                 const unsigned int nwork,
                 const unsigned work_offset,
                 const unsigned int maxn,
                 bool patch,
                 const unsigned int *d_nlist_patch_old,
                 const unsigned int *d_nlist_patch_new,
                 const unsigned int *d_nneigh_patch_old,
                 const unsigned int *d_nneigh_patch_new,
                 const float *d_energy_old,
                 const float *d_energy_new,
                 const unsigned int maxn_patch,
                 unsigned int *d_condition,
                 const unsigned int seed,
                 const unsigned int select,
                 const unsigned int timestep)
    {
    unsigned offset = threadIdx.x;
    unsigned int group_size = blockDim.x;
    unsigned int group = threadIdx.y;
    unsigned int n_groups = blockDim.y;
    bool master = offset == 0;

    // the particle we are handling
    unsigned int i = blockIdx.x*n_groups + group;
    bool active = true;
    if (i >= nwork)
        active = false;
    i += work_offset;

    extern __shared__ char sdata[];

    float *s_energy_old = (float *) sdata;
    float *s_energy_new = (float *) (s_energy_old + n_groups);
    unsigned int *s_reject = (unsigned int *) (s_energy_new + n_groups);

    bool move_active = false;
    if (active && master)
        {
        s_reject[group] = d_reject_out_of_cell[i];
        s_energy_old[group] = 0.0f;
        s_energy_new[group] = 0.0f;
        }

    if (active)
        {
        move_active = d_trial_move_type[i] > 0;
        }

    __syncthreads();

    if (active && move_active)
        {
        unsigned int update_order_i = d_update_order_by_ptl[i];

        // iterate over overlapping neighbors in old configuration
        unsigned int nneigh = d_nneigh[i];
        bool accept = true;
        for (unsigned int cur_neigh = offset; cur_neigh < nneigh; cur_neigh += group_size)
            {
            unsigned int primitive = d_nlist[cur_neigh+maxn*i];

            unsigned int j = primitive;
            bool old = true;
            if (j >= N_old)
                {
                j -= N_old;
                old = false;
                }

            // has j been updated? ghost particles are not updated
            bool j_has_been_updated = j < N && d_trial_move_type[j]
                && d_update_order_by_ptl[j] < update_order_i && !d_reject[j];

            // acceptance, reject if current configuration of particle overlaps
            if ((old && !j_has_been_updated) || (!old && j_has_been_updated))
                {
                accept = false;
                break;
                }

            } // end loop over neighbors

        if (!accept)
            {
            atomicMax(&s_reject[group], 1);
            }

        if (patch)
            {
            // iterate over overlapping neighbors in old configuration
            float energy_old = 0.0f;
            unsigned int nneigh = d_nneigh_patch_old[i];
            bool evaluated = false;
            for (unsigned int cur_neigh = offset; cur_neigh < nneigh; cur_neigh += group_size)
                {
                unsigned int primitive = d_nlist_patch_old[cur_neigh+maxn_patch*i];

                unsigned int j = primitive;
                bool old = true;
                if (j >= N_old)
                    {
                    j -= N_old;
                    old = false;
                    }

                // has j been updated? ghost particles are not updated
                bool j_has_been_updated = j < N && d_trial_move_type[j]
                    && d_update_order_by_ptl[j] < update_order_i && !d_reject[j];

                if ((old && !j_has_been_updated) || (!old && j_has_been_updated))
                    {
                    energy_old += d_energy_old[cur_neigh+maxn_patch*i];
                    evaluated = true;
                    }

                } // end loop over neighbors

            if (evaluated)
                atomicAdd(&s_energy_old[group], energy_old);

            // iterate over overlapping neighbors in new configuration
            float energy_new = 0.0f;
            nneigh = d_nneigh_patch_new[i];
            evaluated = false;
            for (unsigned int cur_neigh = offset; cur_neigh < nneigh; cur_neigh += group_size)
                {
                unsigned int primitive = d_nlist_patch_new[cur_neigh+maxn_patch*i];

                unsigned int j = primitive;
                bool old = true;
                if (j >= N_old)
                    {
                    j -= N_old;
                    old = false;
                    }

                // has j been updated? ghost particles are not updated
                bool j_has_been_updated = j < N && d_trial_move_type[j]
                    && d_update_order_by_ptl[j] < update_order_i && !d_reject[j];

                if ((old && !j_has_been_updated) || (!old && j_has_been_updated))
                    {
                    energy_new += d_energy_new[cur_neigh+maxn_patch*i];
                    evaluated = true;
                    }

                } // end loop over neighbors

            if (evaluated)
                atomicAdd(&s_energy_new[group], energy_new);
            }
        } // end if (active && move_active)

    __syncthreads();

    if (master && active && move_active)
        {
        float delta_U = s_energy_new[group] - s_energy_old[group];

        // Metropolis-Hastings
        hoomd::RandomGenerator rng_i(hoomd::RNGIdentifier::HPMCMonoAccept, seed, i, select, timestep);
        bool accept = !s_reject[group] && (!patch || (hoomd::detail::generate_canonical<double>(rng_i) < slow::exp(-delta_U)));

        if ((accept && d_reject[i]) || (!accept && !d_reject[i]))
            {
            // flag that we're not done yet (a trivial race condition upon write)
            *d_condition = 1;
            }

        // write out to device memory
        d_reject_out[i] = accept ? 0 : 1;
        }
    }

} // end namespace kernel

//! Driver for kernel::hpmc_excell()
void hpmc_excell(unsigned int *d_excell_idx,
                 unsigned int *d_excell_size,
                 const Index2D& excli,
                 const unsigned int *d_cell_idx,
                 const unsigned int *d_cell_size,
                 const unsigned int *d_cell_adj,
                 const Index3D& ci,
                 const Index2D& cli,
                 const Index2D& cadji,
                 const unsigned int ngpu,
                 const unsigned int block_size)
    {
    assert(d_excell_idx);
    assert(d_excell_size);
    assert(d_cell_idx);
    assert(d_cell_size);
    assert(d_cell_adj);

    // determine the maximum block size and clamp the input block size down
    static int max_block_size = -1;
    if (max_block_size == -1)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::hpmc_excell)));
        max_block_size = attr.maxThreadsPerBlock;
        }

    // setup the grid to run the kernel
    dim3 threads(min(block_size, (unsigned int)max_block_size), 1, 1);
    dim3 grid(ci.getNumElements() / block_size + 1, 1, 1);

    hipLaunchKernelGGL(kernel::hpmc_excell, dim3(grid), dim3(threads), 0, 0, d_excell_idx,
                                           d_excell_size,
                                           excli,
                                           d_cell_idx,
                                           d_cell_size,
                                           d_cell_adj,
                                           ci,
                                           cli,
                                           cadji,
                                           ngpu);

    }

//! Kernel driver for kernel::hpmc_shift()
void hpmc_shift(Scalar4 *d_postype,
                int3 *d_image,
                const unsigned int N,
                const BoxDim& box,
                const Scalar3 shift,
                const unsigned int block_size)
    {
    assert(d_postype);
    assert(d_image);

    // setup the grid to run the kernel
    dim3 threads_shift(block_size, 1, 1);
    dim3 grid_shift(N / block_size + 1, 1, 1);

    hipLaunchKernelGGL(kernel::hpmc_shift, dim3(grid_shift), dim3(threads_shift), 0, 0, d_postype,
                                                      d_image,
                                                      N,
                                                      box,
                                                      shift);

    // after this kernel we return control of cuda managed memory to the host
    hipDeviceSynchronize();
    }


void hpmc_accept(const unsigned int *d_update_order_by_ptl,
                 const unsigned int *d_trial_move_type,
                 const unsigned int *d_reject_out_of_cell,
                 unsigned int *d_reject,
                 unsigned int *d_reject_out,
                 const unsigned int *d_nneigh,
                 const unsigned int *d_nlist,
                 const unsigned int N_old,
                 const unsigned int N,
                 const GPUPartition& gpu_partition,
                 const unsigned int maxn,
                 bool patch,
                 const unsigned int *d_nlist_patch_old,
                 const unsigned int *d_nlist_patch_new,
                 const unsigned int *d_nneigh_patch_old,
                 const unsigned int *d_nneigh_patch_new,
                 const float *d_energy_old,
                 const float *d_energy_new,
                 const unsigned int maxn_patch,
                 unsigned int *d_condition,
                 const unsigned int seed,
                 const unsigned int select,
                 const unsigned int timestep,
                 const unsigned int block_size,
                 const unsigned int tpp)
    {
    // determine the maximum block size and clamp the input block size down
    static int max_block_size = -1;
    if (max_block_size == -1)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::hpmc_accept)));
        max_block_size = attr.maxThreadsPerBlock;
        }

    // setup the grid to run the kernel
    unsigned int run_block_size = min(block_size, (unsigned int)max_block_size);

    // threads per particle
    unsigned int cur_tpp = min(run_block_size,tpp);
    while (run_block_size % cur_tpp != 0)
        cur_tpp--;

    unsigned int n_groups = run_block_size/cur_tpp;
    dim3 threads(cur_tpp, n_groups, 1);

    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;
        const unsigned int num_blocks = (nwork + n_groups - 1)/n_groups;
        dim3 grid(num_blocks, 1, 1);

        unsigned int shared_bytes = n_groups * (2*sizeof(float) + sizeof(unsigned int));
        hipLaunchKernelGGL(kernel::hpmc_accept, grid, threads, shared_bytes, 0,
            d_update_order_by_ptl,
            d_trial_move_type,
            d_reject_out_of_cell,
            d_reject,
            d_reject_out,
            d_nneigh,
            d_nlist,
            N_old,
            N,
            nwork,
            range.first,
            maxn,
            patch,
            d_nlist_patch_old,
            d_nlist_patch_new,
            d_nneigh_patch_old,
            d_nneigh_patch_new,
            d_energy_old,
            d_energy_new,
            maxn_patch,
            d_condition,
            seed,
            select,
            timestep);
        }
    }

} // end namespace gpu
} // end namespace hpmc

